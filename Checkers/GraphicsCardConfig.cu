#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "GraphicsCardConfig.h"

namespace Checkers
{

int GetSmCount()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	return prop.multiProcessorCount;
}

int GetThreadsPerSm()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    switch (prop.major)
    {
    case 6:
        return prop.minor == 0 ? 64 : 128;
    case 7:
        return 64;
    case 8:
        return prop.minor == 0 ? 64 : 128;
    case 9:
        return 128;
    default:
        return 64;
    }

}

}