#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <thrust/reduce.h>

#include <stdio.h>

#include <iostream>
#include <random>

#include "Core/Core.h"

#include "DeviceSimulator.h"

namespace Checkers
{

using CudaAssert = Assert<hipError_t, hipGetLastError, hipSuccess, hipGetErrorString>;

__device__ DeviceGenerator::DeviceGenerator(unsigned int seed, unsigned int scramble)
{
	hiprand_init(seed, scramble, 0, &m_State);
}

__device__ unsigned int DeviceGenerator::GetUniform(unsigned int low, unsigned int high)
{
	return low + (1.0f - hiprand_uniform(&m_State)) * high;
}

static __global__ void SetupKernel(unsigned int seed, DeviceGenerator *generators)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	// Placement new - the memory is already allocated
	// we just want to run the constructor to initialize the object in place
	new (&generators[tid]) DeviceGenerator(seed, tid);
}

DeviceSimulator::DeviceSimulator(unsigned int blockCount, unsigned int threadsPerBlock)
	: m_BlockCount(blockCount), m_ThreadsPerBlock(threadsPerBlock), m_ThreadCount(m_BlockCount* m_ThreadsPerBlock)
{
	std::random_device dev;
	unsigned int seed = dev();

	hipMalloc(&m_dPositions, sizeof(Position) * m_BlockCount);
	hipMalloc(&m_dBlackInc, sizeof(int) * m_BlockCount);
	hipMalloc(&m_dWhiteInc, sizeof(int) * m_BlockCount);

	hipMalloc(&m_Generators, sizeof(DeviceGenerator) * m_ThreadCount);
	SetupKernel<<<m_BlockCount, m_ThreadsPerBlock>>>(seed, m_Generators);
	hipDeviceSynchronize();
}

DeviceSimulator::~DeviceSimulator()
{
	// ShutdownKernel invoking the destructors of generators should go here
	hipFree(m_Generators);

	hipFree(&m_dPositions);
	hipFree(&m_dBlackInc);
	hipFree(&m_dWhiteInc);
}

static __global__ void SimulateKernel(Position *positions, DeviceGenerator *generators, int *blackInc, int *whiteInc)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	static constexpr int MaxBlockSize = 1024;
	__shared__ int biSum[MaxBlockSize];
	__shared__ int wiSum[MaxBlockSize];

	if (threadIdx.x == 0)
	{
		biSum[blockIdx.x] = 0;
		wiSum[blockIdx.x] = 0;
	}

	Position position = positions[blockIdx.x];
	position.SimulateOne(generators[tid], biSum[threadIdx.x], wiSum[threadIdx.x]);
	__syncthreads();

#pragma unroll
	for (int i = 1; i <= 10; i++)
	{
		const int activeThreads = (blockDim.x >> i);
		if (threadIdx.x < activeThreads)
		{
			biSum[threadIdx.x] += biSum[activeThreads + threadIdx.x];
			wiSum[threadIdx.x] += wiSum[activeThreads + threadIdx.x];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		blackInc[blockIdx.x] = biSum[0];
		whiteInc[blockIdx.x] = wiSum[0];
	}
}

void DeviceSimulator::Simulate(const std::vector<Position> &positions, std::vector<int> &blackInc, std::vector<int> &whiteInc, std::vector<int> &visitsInc)
{
	int blockCount = positions.size();
	if (blockCount > m_BlockCount)
		blockCount = m_BlockCount;

	hipMemcpy(m_dPositions, positions.data(), sizeof(Position) * blockCount, hipMemcpyHostToDevice);

	{
		Timer timer("Kernel");
		SimulateKernel<<<blockCount, m_ThreadsPerBlock>>>(m_dPositions, m_Generators, m_dBlackInc, m_dWhiteInc);
		hipDeviceSynchronize();
	}

	hipMemcpy(blackInc.data(), m_dBlackInc, sizeof(int) * blockCount, hipMemcpyDeviceToHost);
	hipMemcpy(whiteInc.data(), m_dWhiteInc, sizeof(int) * blockCount, hipMemcpyDeviceToHost);
	std::fill(visitsInc.begin(), visitsInc.begin() + blockCount, m_ThreadsPerBlock * 2);

	std::fill(blackInc.begin() + blockCount, blackInc.end(), 0);
	std::fill(whiteInc.begin() + blockCount, whiteInc.end(), 0);
	std::fill(visitsInc.begin() + blockCount, visitsInc.end(), 0);
}

}
